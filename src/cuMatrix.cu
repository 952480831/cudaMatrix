#include "hip/hip_runtime.h"
#include "cuMatrix.hu"

//! Populate the scope with current position
#define ESTABLISH_CURRENT_POSITION unsigned int y, x, pos; \
	y = blockIdx.y*blockDim.y + threadIdx.y; \
	x = blockIdx.x*blockDim.x + threadIdx.x; \
	pos = y*width+x;

__global__ void _matDecomposeLU(float* src, float *l, float *u, unsigned int width)
{
	unsigned int i;
	ESTABLISH_CURRENT_POSITION;

	if(x<y) //we are in the L part
	{
		u[pos] = 0.;
		l[pos] = src[pos];
		for(i=0; i<x; i++)
			l[pos] -= src[y*width+i]*src[i*width+x];
		l[pos] /= src[x*width+x];
		return;
	}

	//we are in the U part
	if(x==y)
		l[pos] = 1.;
	else l[pos] = 0.;

	u[pos] = src[pos];
	for(i=0; i<x; i++)
		u[pos] -= src[y*width+i]*src[i*width+x];
}

__global__ void _matMultiply(float *a, float *b, float *dest, unsigned int width)
{
	unsigned int i;
	ESTABLISH_CURRENT_POSITION;

	dest[pos] = a[y*width] * b[x];
	for(i=1; i<width; i++)
		dest[pos] += a[y*width+i]*b[i*width+x];
}
